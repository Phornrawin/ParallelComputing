#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

__global__ void reduceInterleaved(int *d_A, int n) {
	int id = threadIdx.x;
	int s = 1;
	for (int i = 0; i < (int)log2((double)n); i++){
		if (id < n / 2) {
			d_A[(int)pow((double)2, (double)(i + 1)) * id] = d_A[(int)pow((double)2, (double)(i + 1)) * id] + d_A[(int)pow((double)2, (double)(i + 1)) * id + s];
			s = s * 2;
		}
		__syncthreads();
	}
}

__global__ void reduceContiguous(int *d_A, int n) {
	int id = threadIdx.x;
	int s = n/2;
	for (int i = 0; i < (int)log2((double)n); i++) {
		if (id < n / 2) {
			d_A[id] = d_A[id] + d_A[id + s];
			s = s / 2;
		}
		__syncthreads();
	}
}

int main()
{
	const int N = 64;
	int h_A[N];
	for (int i = 0; i < N; i++) {
		h_A[i] = i;
	}
	

	int* d_A;
	hipMalloc((void **)&d_A, sizeof(int)*(N));
	hipMemcpy(d_A, &h_A, sizeof(int)*(N), hipMemcpyHostToDevice);

	//define DUDA Timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//start Cuda timer
	hipEventRecord(start, 0);


	//stencil<<<1, 1 >>> (d_in, d_out, N, k);  
	//fast_stencil <<<blockNum, blockSize>>> (d_in, d_out, N, k);
	reduceContiguous<< <1, 1024 >> > (d_A, N);


	//stop Cuda timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	//compute elapsed time
	float time;
	hipEventElapsedTime(&time, start, stop);


	hipMemcpy(&h_A, d_A, N * sizeof(int), hipMemcpyDeviceToHost);


	
	printf("Out[0] = %d \n",h_A[0]);


	hipFree(d_A);

	//report time in kernel
	printf("Time in kernel = %f ms \n", time);



	return 0;
}

// Helper function for using CUDA to add vectors in parallel.


