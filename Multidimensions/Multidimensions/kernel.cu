#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <math.h>
#include <stdio.h>

__global__ void sum_matrix(int* d_x, int* d_y, int* d_z, int n) {
	int x = threadIdx.x;
	int y = threadIdx.y;
	d_z[x + y] = d_x[x + y] + d_y[x + y];	
}

int main()
{
	const int N = 10;
	int* h_x = new int[N*N];
	int* h_y = new int[N*N];
	int* h_z = new int[N*N];
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			h_x[i] = i;
			h_y[i] = i;
			h_z[i] = 0;
		}
	}

	int* d_x;
	int* d_y;
	int* d_z;
	hipMalloc((void **)&d_x, sizeof(int)*(N*N));
	hipMalloc((void **)&d_y, sizeof(int)*(N*N));
	hipMalloc((void **)&d_z, sizeof(int)*(N*N));
	hipMemcpy(d_x, &h_x, sizeof(int)*(N*N), hipMemcpyHostToDevice);
	hipMemcpy(d_y, &h_y, sizeof(int)*(N*N), hipMemcpyHostToDevice);
	hipMemcpy(d_z, &h_z, sizeof(int)*(N*N), hipMemcpyHostToDevice);

	//define DUDA Timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//start Cuda timer
	hipEventRecord(start, 0);


	//stencil<<<1, 1 >>> (d_in, d_out, N, k);  
	//fast_stencil <<<blockNum, blockSize>>> (d_in, d_out, N, k);
	dim3 dimGrid(1, 1, 1);
	dim3 dimBlock(10, 10, 1);
	sum_matrix<< <dimGrid, dimBlock>> > (d_x, d_y, d_z, N);


	//stop Cuda timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	//compute elapsed time
	float time;
	hipEventElapsedTime(&time, start, stop);


	hipMemcpy(&h_x, d_x, N*N * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_y, d_x, N*N * sizeof(int), hipMemcpyDeviceToHost);
	

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
		}
		printf("Out[%d, %d] = %d \n", i, j, h_z[i]);	
	}

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);

	//report time in kernel
	printf("Time in kernel = %f ms \n", time);



	delete[] h_x;
	delete[] h_y;
	delete[] h_z;

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.


